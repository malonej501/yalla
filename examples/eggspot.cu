#include "hip/hip_runtime.h"
// Toy model for accessing cell fate decisions during cancer development

// Compilation
//
// $ nvcc -std=c++14 -arch=sm_86 {"compiler flags"} Limb_model_simulation.cu
// The values for "-std" and "-arch" flags will depend on your version of CUDA and the specific GPU model you have respectively.
// e.g. -std=c++14 works for CUDA version 11.6 and -arch=sm_86 corresponds to the generation of NVIDIA Geforce 30XX cards.
#include "../include/solvers.cuh"
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"  
#include "../include/property.cuh"
#include "../include/utils.cuh"
#include "../include/vtk.cuh"

// N.B. distances are in millimeters so 0.001 = 1 micrometer

// global simulation parameters
const float r_max = 0.1;                        // Max distance betwen two cells for which they will interact - set to upper bound of donut
const int n_max = 200000;                       // Max number of cells
const float noise = 0;//0.5;                        // Magnitude of noise returned by generate_noise
const int cont_time = 1000;                    // Simulation duration in arbitrary time units 1 = 1 day
const float dt = 0.1;                           // Time step for Euler integration
const int no_frames = 100;                      // no. frames of simulation output to vtk - divide the simulation time by this number

// tissue initialisation
const float init_dist = 0.05;//0.082;                    // mean distance between cells when initialised 
const float div_dist = 0.01;
const int n_0 = 1000;//450;//500;//350;                            // Initial number of cells n.b. this number needs to divide properly between stripes if using volk initial condition
const float A_init = 0;                         // % of the initial cell population that will be type 1 / A

// cell migration parameters
const bool diff_adh_rep = true;                // set to false to turn off differential adhesion and repulsion
// const float rii = 0.02;                         // Length scales for migration forces for iri-iri (in mm)
// const float Rii = 0.00124;                      // Repulsion from iri to iri (mm^2/day)
// const float aii = 0.012;
// const float Aii = 0.001956;

// const float rii = 0.01;                         // Length scales for migration forces for iri-iri (in mm)
// const float Rii = 0.002;                      // Repulsion from iri to iri (mm^2/day)
// const float aii = 0.011;
// const float Aii = 0.0019;

const float rii = 0.012;                         // Length scales for migration forces for iri-iri (in mm)
const float Rii = 0.0045;                      // Repulsion from iri to iri (mm^2/day)
const float aii = 0.019;
const float Aii = 0.0019;

// proliferation parameters
const float A_div = 0.012;                      // 0.02 works well if you have the overcrowding condition
const float B_div = 0.012;                   
const float r_A_birth = 0.08;                   //chance of iridophore birth from background cell
const int Acrowd = 5;                           // max no. A cells in the local disc of A cells before proliferation stops
const int Bcrowd = 5;                           // max no. B cells in the local disc of B cells before proliferation stops
const float uthresh = 0.015;                      // B cells will not change to A if the amount of u exceeds this value

// chemical diffusion rates - this is Fick's first law?
const float D_u = 1.0;
const float D_v = 0.01;

// Macro that builds the cell variable type - instead of type float3 we are making a instance of Cell with attributes x,y,z,u,v where u and v are diffusible chemicals
//MAKE_PT(Cell); // float3 i .x .y .z .u .v .whatever
// to use MAKE_PT(Cell) replace every instance of float3 with Cell
MAKE_PT(Cell, u, v);

__device__ float* d_mechanical_strain; // define global variable for mechanical strain on the GPU (device)
__device__ int* d_cell_type; // global variable for cell type on the GPU - iridophore=1, xanthophore=2, DEAD=0
__device__ Cell* d_W; // global variable for random number from Weiner process for stochasticity
__device__ int* d_ngs_type_A; // no. iri cells in neighbourhood
__device__ int* d_ngs_type_B; // no. xan cells in neighbourhood

template<typename Pt>
__device__ Pt pairwise_force(Pt Xi, Pt r, float dist, int i, int j)
{
    Pt dF{0};

    //if (dist > r_max) return dF; // Gabriel solver doesn't account for distance when computing neighbourhood, we need to exclude distant pairs
    if (dist > r_max) return dF; // set cutoff for computing forces


    // This will be only useful in simulations with a wall and a ghost node
    if (i == j){
        dF += d_W[i]; // add stochasticity from the weiner process to the attributes of the cells

        // each cell type has a base line production rate of chemical u or v depending on cell type
        float k_prod = 0.3;
        // dF.u += k_prod * (d_cell_type[i] == 1); // cell type 1 produces chemical u
        // dF.v += k_prod * (d_cell_type[i] == 2); // cell type 2 produces chemical v
        dF.u = k_prod * (1.0 - Xi.u) * (d_cell_type[i] ==1);
        dF.v = k_prod * (1.0 - Xi.v) * (d_cell_type[i] ==2);


        // add degredation not dependent on anything
        float k_deg = 0.03;
        dF.u -= k_deg * (Xi.u);
        dF.v -= k_deg * (Xi.v);

        return dF;
    }
    
    dF.u = -D_u * r.u; // r.u is the difference in chemical concentration between cells in pair
    dF.v = -D_v * r.v;


    if (dist < 0.075) { // the radius of the inner disc
        // count no. each cell type in neighbourhood
        if (d_cell_type[j] == 1) d_ngs_type_A[i] += 1;
        else d_ngs_type_B[i] += 1;
    }



    // we define the default strength of adhesion and repulsion
    float Adh = 0;
    float adh = 1;
    float Rep = Rii;
    float rep = rii;

    if (diff_adh_rep) {
        if (d_cell_type[i] == 1 and d_cell_type[j] == 1) { // iri -> iri
            Adh = Aii;
            adh = aii;
            Rep = Rii;
            rep = rii;
        }
    }


    // float F = (k_rep * fmaxf(0.08 - dist, 0) - k_adh * fmaxf(dist - 0.08, 0)); // forces are also dependent on adhesion and repulsion between cell types
    // float F = (Adh * r.x * exp(-sqrt(r.x^2 + r.y^2) / adh)) / (adh * sqrt(r.x^2 + r.y^2)) - (Rep * r.x * exp(-sqrt(r.x^2 - r.y^2) / rep) / (rep * sqrt(r.x^2 - r.y^2)));
    // Volkening et al. 2015 force potential, function in terms of distance in n dimensions
    float term1 = Adh/adh * expf(-dist / adh);
    float term2 = Rep/rep * expf(-dist / rep);
    float F = term1 - term2;
    //printf("%f\n", F);
    d_mechanical_strain[i] -= F; // mechanical strain is the sum of forces on the cell

    dF.x -= r.x * F / dist;
    dF.y -= r.y * F / dist;
    dF.z -= 0;  


    return dF;
}

__global__ void generate_noise(int n, hiprandState* d_state) { // Weiner process for Heun's method
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float D = noise; // the magnitude of random noise - set to 0 for deterministic simulation

    // return noise for every attribute of the cell in this case x,y,z
    d_W[i].x = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].y = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    //d_W[i].z = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].z = 0;
    d_W[i].u = 0;
    d_W[i].v = 0;
}

__global__ void proliferation(int n_cells, hiprandState* d_state, Cell* d_X, float3* d_old_v, int* d_n_cells) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // get the index of the current cell
    if (i >= n_cells) return; // return nothing if the index is greater than n_cells
    if (n_cells >= (n_max * 0.9)) return;  // return nothing if the no. cells starts to approach the max

    if (d_cell_type[i] == 1) {
        // if (d_ngs_type_A[i] + d_ngs_type_B[i] > eta) return;
        if (d_ngs_type_A[i] > Acrowd) return;
        if (hiprand_uniform(&d_state[i]) > (A_div * dt)) return;
    }

    if (d_cell_type[i] == 2) {
        //if (d_ngs_type_A[i] + d_ngs_type_B[i] < eta) return;
        // if (d_ngs_type_A[i] + d_ngs_type_B[i] > eta) return;
        //if (d_ngs_type_A[i] + d_ngs_type_B[i] > ABcrowd) return;
        if (d_ngs_type_B[i] > Bcrowd) return;
        if (hiprand_uniform(&d_state[i]) > (B_div * dt)) return;
    }
    

    int n = atomicAdd(d_n_cells, 1);

    // new cell added next to parent at random angle
    float theta = acosf(2. * hiprand_uniform(&d_state[i]) - 1);
    float phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;

    d_X[n].x = d_X[i].x + div_dist / 2 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + div_dist / 2 * sinf(theta) * sinf(phi);
    d_X[n].z = 0;

    d_old_v[n] = d_old_v[i];

    d_mechanical_strain[n] = 0.0;
     
    
    // set child cell types    
    if (d_cell_type[i] == 2) {
        d_cell_type[n] = (hiprand_uniform(&d_state[i]) < r_A_birth and d_X[i].u < uthresh) ? 1 : 2; // sometimes cell type 2 produces cell type 1 random birth of cell type 1 is inhibited by chemical u
    }
    if (d_cell_type[i] == 1) {
        d_cell_type[n] = 1;
    }

    // set child cell chemical amounts
    // d_X[n].u = (d_cell_type[n] == 1) ? 1 : 0;     // if the child is type 1, it is given u=1,v=0 if not, u=0,v=1
    // d_X[n].v = (d_cell_type[n] == 1) ? 0 : 1;

     // half the amount of each chemical upon cell division in the parent cell
    d_X[i].u *= 0.5;
    d_X[i].v *= 0.5;
    // the child inherits the other half of the amount of the chemical
    d_X[n].u = d_X[i].u;
    d_X[n].v = d_X[i].v;
 
}


int main(int argc, char const* argv[])
{

    std::cout << std::fixed << std::setprecision(6); // set precision for floats

    // Print the parameters
    std::cout << "Global Simulation Parameters:\n";
    std::cout << "r_max = " << r_max << "\n";
    std::cout << "n_max = " << n_max << "\n";
    std::cout << "noise = " << noise << "\n";
    std::cout << "cont_time = " << cont_time << "\n";
    std::cout << "dt = " << dt << "\n";
    std::cout << "no_frames = " << no_frames << "\n\n";

    std::cout << "Tissue Initialization:\n";
    std::cout << "init_dist = " << init_dist << "\n";
    std::cout << "div_dist = " << div_dist << "\n";
    std::cout << "n_0 = " << n_0 << "\n";
    std::cout << "A_init = " << A_init << "\n\n";

    std::cout << "Cell Migration Parameters:\n";
    std::cout << "diff_adh_rep = " << (diff_adh_rep ? "true" : "false") << "\n";
    std::cout << "rii = " << rii << "\n";
    std::cout << "Rii = " << Rii << "\n";
    std::cout << "aii = " << aii << "\n";
    std::cout << "Aii = " << Aii << "\n\n";

    std::cout << "Proliferation Parameters:\n";
    std::cout << "A_div = " << A_div << "\n";
    std::cout << "B_div = " << B_div << "\n";
    std::cout << "r_A_birth = " << r_A_birth << "\n";
    std::cout << "Acrowd = " << Acrowd << "\n";
    std::cout << "Bcrowd = " << Bcrowd << "\n";
    std::cout << "uthresh = " << uthresh << "\n\n";

    std::cout << "Chemical Diffusion Rates:\n";
    std::cout << "D_u = " << D_u << "\n";
    std::cout << "D_v = " << D_v << "\n\n";


    /*
    Prepare Random Variable for the Implementation of the Wiener Process
    */
    hiprandState* d_state; // define the random number generator on the GPu
    hipMalloc(&d_state, n_max*sizeof(hiprandState)); // allocate GPU memory according to the number of cells
    auto seed = time(NULL); // random number seed - coupled to the time on your machine
    setup_rand_states<<<(n_max + 32 - 1)/32, 32>>>(n_max, seed, d_state); // configuring the random number generator on the GPU (provided by utils.cuh)

    /* create host variables*/
    // Wiener process
    Property<Cell> W{n_max, "wiener_process"}; // define a property for the weiner process
    hipMemcpyToSymbol(HIP_SYMBOL(d_W), &W.d_prop, sizeof(d_W)); // connect the global property defined on the GPU to the property defined in this function

    // Mechanical strain
    Property<float> mechanical_strain{n_max, "mech_str"}; // create an instance of the property
    hipMemcpyToSymbol(HIP_SYMBOL(d_mechanical_strain), &mechanical_strain.d_prop, sizeof(d_mechanical_strain)); // connect the above instance (on the host) to the global variable on the device

    // No. iri in neighbourhood
    Property<int> ngs_type_A{n_max, "ngs_type_A"}; // create an instance of the property
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_type_A), &ngs_type_A.d_prop, sizeof(d_ngs_type_A));
    // No. xan in neighbourhood
    Property<int> ngs_type_B{n_max, "ngs_type_B"}; // create an instance of the property
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_type_B), &ngs_type_B.d_prop, sizeof(d_ngs_type_B));

    // Cell type labels
    Property<int> cell_type{n_max, "cell_type"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_cell_type), &cell_type.d_prop, sizeof(d_cell_type));

    for (int i =0; i < n_0; i++) {
        cell_type.h_prop[i] = (std::rand() % 100 < A_init) ?  1 : 2; //randomly assign a proportion of initial cells with each type
    }

    // for (int i =0; i < n_0; i++) {
    //     cell_type.h_prop[i] = 2; // set all initial cells to be background
    // }

    
    /**/

    // Initial conditions
    
    Solution<Cell, Gabriel_solver> cells{n_max, 50, r_max};
    // Solution<Cell, Grid_solver> cells{n_max, 50, r_max}; //originally using r_max*5
    *cells.h_n = n_0;
    //random_sphere(0.7, cells);
    random_disk_z(init_dist, cells);
    // regular_rectangle(init_dist, std::round(std::sqrt(n_0) / 10) * 10, cells); //initialise square with nx=n_0/2 center will be at (y,x) = (1,1)

    // initialise chemical amounts 
    for (int i = 0; i < n_0; i++) {
        // cells.h_X[i].u = 0; //h_X is host cell
        // cells.h_X[i].v = 0;
        if (cell_type.h_prop[i] == 1) {
            cells.h_X[i].u = 1;
            cells.h_X[i].v = 0;
        } else if (cell_type.h_prop[i] == 2) {
            cells.h_X[i].u = 0;
            cells.h_X[i].v = 1;
        }
    }
    
    // Initialise properties and k with zeroes
    for (int i = 0; i < n_max; i++) { //initialise with zeroes, for loop step size is set to 1 with i++
        mechanical_strain.h_prop[i] = 0;
        ngs_type_A.h_prop[i] = 0;
        ngs_type_B.h_prop[i] = 0;
    }

    auto generic_function = [&](const int n, const Cell* __restrict__ d_X, Cell* d_dX) { // then set the mechanical forces to zero on the device
        // Set these properties to zero after every timestep so they don't accumulate
        thrust::fill(thrust::device, mechanical_strain.d_prop, mechanical_strain.d_prop + cells.get_d_n(), 0.0);
        thrust::fill(thrust::device, ngs_type_A.d_prop, ngs_type_A.d_prop + cells.get_d_n(), 0);
	    thrust::fill(thrust::device, ngs_type_B.d_prop, ngs_type_B.d_prop + cells.get_d_n(), 0);
    };

    cells.copy_to_device();
    mechanical_strain.copy_to_device();
    cell_type.copy_to_device();
    ngs_type_A.copy_to_device();
    ngs_type_B.copy_to_device();


        
    Vtk_output output{"out"};



    /* the neighbours are initialised with 0. However, you want to use them in the proliferation function, which is called first.
	1. proliferation
	2. noise
	3. take_step
       we use a trick, such that the very first call of the proliferation is not launched on zeros.
       here instead of dt we pass 0.0, so that we count cells, but do not compute any replacements in the tissue
       -> x[t+1] = x[t] + 0.0 * (dx);
    */

    cells.take_step<pairwise_force>(0.0, generic_function);
    
    // write out initial condition
    cells.copy_to_host();
    mechanical_strain.copy_to_host();
    cell_type.copy_to_host();
    ngs_type_A.copy_to_host();
    ngs_type_B.copy_to_host();

    output.write_positions(cells);
    output.write_property(mechanical_strain);
    output.write_property(cell_type);
    output.write_property(ngs_type_A);
    output.write_property(ngs_type_B);
    output.write_field(cells, "u", &Cell::u); //write the u part of each cell to vtk
    output.write_field(cells, "v", &Cell::v);


    // Main simulation loop
    for (int time_step = 0; time_step <= cont_time; time_step ++) {
        for (float T = 0.0; T < 1.0; T+=dt) {
            generate_noise<<<(cells.get_d_n() + 32 - 1)/32, 32>>>(cells.get_d_n(), d_state); // generate random noise which we will use later on to move the cells
            proliferation<<<(cells.get_d_n() + 128 - 1)/128, 128>>>(cells.get_d_n(), d_state, cells.d_X, cells.d_old_v, cells.d_n); // simulate proliferation
            cells.take_step<pairwise_force, friction_on_background>(dt, generic_function);
        }

        if(time_step % int(cont_time / no_frames) == 0){
            cells.copy_to_host();
            mechanical_strain.copy_to_host();
            cell_type.copy_to_host();
            ngs_type_A.copy_to_host();
            ngs_type_B.copy_to_host();

            output.write_positions(cells);
            output.write_property(mechanical_strain);
            output.write_property(cell_type);
            output.write_property(ngs_type_A);
            output.write_property(ngs_type_B);
            output.write_field(cells, "u", &Cell::u); //write the u part of each cell to vtk
            output.write_field(cells, "v", &Cell::v);
        }
    }
    return 0;
}