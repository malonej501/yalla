#include "hip/hip_runtime.h"
// Toy model for accessing cell fate decisions during cancer development

// Compilation
//
// $ nvcc -std=c++14 -arch=sm_86 {"compiler flags"} Limb_model_simulation.cu
// The values for "-std" and "-arch" flags will depend on your version of CUDA and the specific GPU model you have respectively.
// e.g. -std=c++14 works for CUDA version 11.6 and -arch=sm_86 corresponds to the generation of NVIDIA Geforce 30XX cards.

#include "../include/solvers.cuh"
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"  
#include "../include/property.cuh"
#include "../include/utils.cuh"  
#include "../include/vtk.cuh"

const float r_max = 1.2;                        // Max contact distance between cells
const int n_0 = 500;                           // Initial number of cells

const int cont_time = 1000;                  // Simulation duration in arbitrary time units 1000 = 40h ; 750 = 30h
const float dt = 0.1;                           // Time step for Euler integration

// Macro that builds the cell variable type
// MAKE_PT(Cell, u, v); // float3 i .x .y .z .u .v .whatever

__device__ float* d_mechanical_strain; // define global variable for mechanical strain on the GPU (device)
__device__ int* d_cell_type; // global variable for cell type on the GPU
__device__ float3* d_W; // global variable for random number from Weiner process for stochasticity

template<typename Pt>
__device__ Pt pairwise_force(Pt Xi, Pt r, float dist, int i, int j)
{
    Pt dF{0};

    // This will be only useful in simulations with a wall and a ghost node
    if (i == j){
        dF += d_W[i]; // add stochasticity from the weiner process to the attributes of the cells
        return dF;
    }
    if (dist > r_max) return dF;

    float k_adh = (d_cell_type[i] == d_cell_type[j]) ? 3.0 : 1.0; // if the cell types are the same set adhesion to 3.0 if not then 1.0
    float k_rep = (d_cell_type[i] == d_cell_type[j]) ? 1.0 : 3.0;


    float F = (k_adh * fmaxf(0.7 - dist, 0) - k_rep * fmaxf(dist - 0.8, 0)); // forces are also dependent on adhesion and repulsion between cell types
    // printf("%f\n", F);
    d_mechanical_strain[i] += F; // mechanical strain is the sum of forces on the cell

    dF.x += r.x * F / dist;
    dF.y += r.y * F / dist;
    dF.z += r.z * F / dist;

    return dF;
}

__global__ void generate_noise(int n, hiprandState* d_state) { // Weiner process for Heun's method
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float D = 0.4; // the magnitude of random noise - set to 0 for deterministic simulation

    // return noise for every attribute of the cell in this case x,y,z
    d_W[i].x = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].y = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].z = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
}


int main(int argc, char const* argv[])
{

    /*
    Prepare Random Variable for the Implementation of the Wiener Process
    */
    hiprandState* d_state; // define the random number generator on the GPu
    hipMalloc(&d_state, n_0*sizeof(hiprandState)); // allocate GPU memory according to the number of cells
    auto seed = time(NULL); // random number seed - coupled to the time on your machine
    setup_rand_states<<<(n_0 + 32 - 1)/32, 32>>>(n_0, seed, d_state); // configuring the random number generator on the GPU (provided by utils.cuh)

    Property<float3> W{n_0, "wiener_process"}; // define a property for the weiner process
    hipMemcpyToSymbol(HIP_SYMBOL(d_W), &W.d_prop, sizeof(d_W)); // connect the global property defined on the GPU to the property defined in this function

    // Initial conditions
    
    Solution<float3, Gabriel_solver> cells{n_0, 50, r_max};
    *cells.h_n = n_0;
    random_sphere(0.7, cells);

    cells.copy_to_device();

    // Mechanical strain

    Property<float> mechanical_strain{n_0, "mech_str"}; // create an instance of the property
    hipMemcpyToSymbol(HIP_SYMBOL(d_mechanical_strain), &mechanical_strain.d_prop, sizeof(d_mechanical_strain)); // connect the above instance (on the host) to the global variable on the device

    for (int i = 0; i < n_0; i++) { //initialise with zeroes, for loop step size is set to 1 with i++
        mechanical_strain.h_prop[i] = 0;
    }
    mechanical_strain.copy_to_device();

    auto generic_function = [&](const int n, const float3* __restrict__ d_X, float3* d_dX) { // then set the mechanical forces to zero on the device
        thrust::fill(thrust::device, mechanical_strain.d_prop, mechanical_strain.d_prop + cells.get_d_n(), 0.0);
    };

    // Cell type labels

    Property<int> cell_type{n_0, "cell_type"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_cell_type), &cell_type.d_prop, sizeof(d_cell_type));
    for (int i =0; i < n_0; i++) {
        cell_type.h_prop[i] = std::rand() % 2 + 1; // assign each cell randomly the label 1 or 2
    }
    cell_type.copy_to_device();
        
    Vtk_output output{"relaxation"};

    // Main simulation loop
    for (int time_step = 0; time_step <= cont_time; time_step++) {
        for (float T = 0.0; T < 1.0; T+=dt) {
            generate_noise<<<(n_0 + 32 - 1)/32, 32>>>(n_0, d_state); // generate random noise which we will use later on to move the cells
            cells.take_step<pairwise_force, friction_on_background>(dt, generic_function);    
        }

        if(time_step % 10 == 0){
            cells.copy_to_host();
            mechanical_strain.copy_to_host();
            cell_type.copy_to_host();
            output.write_positions(cells);
            output.write_property(mechanical_strain);
            output.write_property(cell_type);
        }
    }
    return 0;
}