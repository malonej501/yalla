#include "hip/hip_runtime.h"
// Simulate mesenchymal intercalation orchestrated by epithelial signals
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/links.cuh"
#include "../include/mesh.cuh"
#include "../include/polarity.cuh"
#include "../include/property.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"


const auto r_max = 1.0;
const auto r_min = 0.8;
const auto dt = 0.1f;
const auto n_max = 150000;
const auto prots_per_cell = 1;
const auto protrusion_strength = 0.2f;
const auto r_protrusion = 2.0f;
const auto mean_proliferation_rate = 0.015f;
const auto n_time_steps = 500;
enum Cell_types { mesenchyme, epithelium };

MAKE_PT(Cell, w, f, theta, phi);


__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;
__device__ int* d_epi_nbs;

__device__ Cell force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) {
        dF.w = -0.01 * (d_type[i] == mesenchyme) * Xi.w;
        if (Xi.w < 0.f) Xi.w = 0.f;
        dF.f = -0.01 * (d_type[i] == mesenchyme) * Xi.f;
        if (Xi.f < 0.f) Xi.f = 0.f;

        return dF;
    }

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.8, 0) * 2.f;
    } else {
        F = fmaxf(0.9 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 2.f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    dF.w = -r.w * (d_type[i] == mesenchyme) * 0.1f;
    dF.f = -r.f * (d_type[i] == mesenchyme) * 0.1f;

    if (d_type[j] == epithelium)
        atomicAdd(&d_epi_nbs[i], 1);
    else
        atomicAdd(&d_mes_nbs[i], 1);

    if (Xi.w < 0.f) Xi.w = 0.f;
    if (Xi.f < 0.f) Xi.f = 0.f;
    if (d_type[i] == mesenchyme or d_type[j] == mesenchyme) return dF;

    dF += bending_force(Xi, r, dist) * 0.15;
    return dF;
}


__global__ void proliferate(float mean_rate, float mean_distance, Cell* d_X,
    float3* d_old_v, int* d_n_cells, hiprandState* d_state)
{
    D_ASSERT(*d_n_cells * mean_rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *d_n_cells * (1 - mean_rate))
        return;  // Dividing new cells is problematic!

    switch (d_type[i]) {
        case mesenchyme: {
            return;  // When changing this, one should use break;
        }
        case epithelium: {
            if (d_epi_nbs[i] > 7) return;

            if (d_mes_nbs[i] < 1) return;

            auto rnd = hiprand_uniform(&d_state[i]);
            if (rnd > mean_rate) return;
        }
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto theta = acosf(2. * hiprand_uniform(&d_state[i]) - 1);
    auto phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    d_X[n].x = d_X[i].x + mean_distance / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance / 4 * sinf(theta) * sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance / 4 * cosf(theta);
    if (d_type[i] == mesenchyme) {
        d_X[n].w = d_X[i].w / 2;
        d_X[i].w = d_X[i].w / 2;
        d_X[n].f = d_X[i].f / 2;
        d_X[i].f = d_X[i].f / 2;
    } else {
        d_X[n].w = d_X[i].w;
        d_X[n].f = d_X[i].f;
    }
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
    d_old_v[n] = d_old_v[i];
}


__global__ void update_protrusions(const int n_cells,
    const Grid* __restrict__ d_grid, const Cell* __restrict d_X,
    hiprandState* d_state, Link* d_link)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells * prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5) / prots_per_cell);
    auto rand_nb_cube =
        d_grid->d_cube_id[j] +
        d_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i]) * 27), 26)];
    auto cells_in_cube =
        d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b =
        d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube] +
                           min(static_cast<int>(
                                   hiprand_uniform(&d_state[i]) * cells_in_cube),
                               cells_in_cube - 1)];
    D_ASSERT(a >= 0);
    D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0);
    D_ASSERT(b < n_cells);
    if (a == b) return;

    if ((d_type[a] != mesenchyme) or (d_type[b] != mesenchyme)) return;

    auto link = &d_link[a * prots_per_cell + i % prots_per_cell];

    auto old_r = d_X[link->a] - d_X[link->b];
    auto old_dist = norm3df(old_r.x, old_r.y, old_r.z);
    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto not_initialized = link->a == link->b;
    auto noise = hiprand_uniform(&d_state[i]);
    auto superficial = d_X[a].w + d_X[b].w > 0.3f;  // sort cells close to the w
    auto parallel_to_w_gradient = false;            // source
    auto normal_to_f_gradient = false;
    if (superficial) {  // cells close to w source align normal to f gradient
        normal_to_f_gradient =
            fabs(new_r.f / new_dist) < fabs(old_r.f / old_dist) * (1.f - noise);
    } else {  // cells far from w source align along w gradient
        parallel_to_w_gradient =
            fabs(new_r.w / new_dist) > fabs(old_r.w / old_dist) * (1.f - noise);
    }

    if (not_initialized or parallel_to_w_gradient or normal_to_f_gradient) {
        link->a = a;
        link->b = b;
    }
}


int main(int argc, char const* argv[])
{
    // Load the initial conditions
    Vtk_input input{"examples/sphere_ic.vtk"};
    int n_0 = input.n_points;
    Solution<Cell, Grid_solver> cells{n_max};
    *cells.h_n = n_0;

    input.read_positions(cells);
    input.read_polarity(cells);

    Property<int> intype{n_max};
    input.read_property(intype, "cell_type");  // read as int, then
    Property<Cell_types> type{n_max};          // translate to Cell_types
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));

    for (int i = 0; i < n_0; i++) {
        cells.h_X[i].w = 0.0f;
        if (intype.h_prop[i] == 0) {
            type.h_prop[i] = mesenchyme;
        } else if (intype.h_prop[i] == 1) {
            type.h_prop[i] = epithelium;
            if (cells.h_X[i].z > 0.0f) {
                cells.h_X[i].w = 1.0f;
                if (cells.h_X[i].x > 0.0f and abs(cells.h_X[i].y) < 2.5f and
                    cells.h_X[i].z < 3.0f)
                    cells.h_X[i].f = 1.0f;
            }
        }
    }
    cells.copy_to_device();
    type.copy_to_device();

    Property<int> n_mes_nbs{n_max, "n_mes_nbs"};
    Property<int> n_epi_nbs{n_max, "n_epi_nbs"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    Links protrusions{n_max * prots_per_cell, protrusion_strength};
    protrusions.set_d_n(n_0 * prots_per_cell);
    auto intercalation = [&](const int n, const Cell* __restrict__ d_X, Cell* d_dX) {
        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);
        return link_forces(protrusions, d_X, d_dX);
    };
    Grid grid{n_max};

    Vtk_output output{"intercalation_w_gradient"};
    for (auto time_step = 0; time_step <= n_time_steps; time_step++) {
        cells.copy_to_host();
        protrusions.copy_to_host();
        type.copy_to_host();

        protrusions.set_d_n(cells.get_d_n() * prots_per_cell);
        grid.build(cells, r_protrusion);
        update_protrusions<<<(protrusions.get_d_n() + 32 - 1) / 32, 32>>>(
            cells.get_d_n(), grid.d_grid, cells.d_X, protrusions.d_state,
            protrusions.d_link);

        cells.take_step<force>(dt, intercalation);

        proliferate<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
            mean_proliferation_rate, r_min, cells.d_X, cells.d_old_v, cells.d_n,
            protrusions.d_state);

        output.write_positions(cells);
        output.write_links(protrusions);
        output.write_property(type);
        output.write_field(cells);
        output.write_field(cells, "f", &Cell::f);
    }

    return 0;
}
