#include "hip/hip_runtime.h"
// Simulate mesenchymal intercalation orchestrated by epithelial signals
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/links.cuh"
#include "../include/mesh.cuh"
#include "../include/polarity.cuh"
#include "../include/property.cuh"
#include "../include/solvers.cuh"
#include "../include/vtk.cuh"


const auto r_max = 1.0;
const auto r_min = 0.8;
const auto dt = 0.1f;
const auto n_0 = 200;
const auto n_max = 4000;
const auto prots_per_cell = 1;
const auto protrusion_strength = 0.25f;
const auto r_protrusion = 2.0f;
const auto proliferation_rate = 0.040f;
enum Cell_types { mesenchyme, epithelium };

MAKE_PT(Cell, w, theta, phi);


__device__ Cell_types* d_type;
__device__ int* d_mes_nbs;
__device__ int* d_epi_nbs;

__device__ Cell force(Cell Xi, Cell r, float dist, int i, int j)
{
    Cell dF{0};

    if (i == j) {
        dF.w = -0.01f * (d_type[i] == mesenchyme) * Xi.w;
        if (Xi.w < 0.0f) dF.w = 0.0f;

        return dF;
    }

    if (dist > r_max) return dF;

    float F;
    if (d_type[i] == d_type[j]) {
        if (d_type[i] == mesenchyme)
            F = fmaxf(0.7 - dist, 0) * 3.0f - fmaxf(dist - 0.8, 0);
        else
            F = fmaxf(0.7 - dist, 0) * 2.0f - fmaxf(dist - 0.8, 0);
    } else {
        F = fmaxf(0.8 - dist, 0) * 2.f - fmaxf(dist - 0.9, 0) * 1.5f;
    }
    dF.x = r.x * F / dist;
    dF.y = r.y * F / dist;
    dF.z = r.z * F / dist;

    dF.w = -r.w * (d_type[i] == mesenchyme) * 0.4f;
    if (Xi.w < 0.f) dF.w = 0.f;

    if (d_type[j] == mesenchyme)
        d_mes_nbs[i] += 1;
    else
        d_epi_nbs[i] += 1;

    if (d_type[i] == mesenchyme or d_type[j] == mesenchyme) return dF;

    dF += bending_force(Xi, r, dist) * 0.10;
    return dF;
}


__global__ void proliferate(float rate, float mean_distance, Cell* d_X,
    float3* d_old_v, int* d_n_cells, hiprandState* d_state)
{
    D_ASSERT(*d_n_cells * rate <= n_max);
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= *d_n_cells * (1 - rate))
        return;  // Dividing new cells is problematic!

    if(d_type[i] == mesenchyme) {
        auto rnd = hiprand_uniform(&d_state[i]);
        if (rnd > rate) return;
    }else{
        if (d_epi_nbs[i] > 14) return;
        if (d_mes_nbs[i] < 1) return;

        auto rnd = hiprand_uniform(&d_state[i]);
        if (rnd > 2.0f * rate) return;
    }

    auto n = atomicAdd(d_n_cells, 1);
    auto theta = acosf(2. * hiprand_uniform(&d_state[i]) - 1);
    auto phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;
    d_X[n].x = d_X[i].x + mean_distance / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + mean_distance / 4 * sinf(theta) * sinf(phi);
    d_X[n].z = d_X[i].z + mean_distance / 4 * cosf(theta);
    if (d_type[i] == mesenchyme) {
        d_X[n].w = d_X[i].w / 2;
        d_X[i].w = d_X[i].w / 2;
    } else {
        d_X[n].w = d_X[i].w;
    }
    d_X[n].theta = d_X[i].theta;
    d_X[n].phi = d_X[i].phi;
    d_type[n] = d_type[i];
    d_old_v[n] = d_old_v[i];
}


__global__ void update_protrusions(const int n_cells,
    const Grid* __restrict__ d_grid, const Cell* __restrict d_X,
    hiprandState* d_state, Link* d_link)
{
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells * prots_per_cell) return;

    auto j = static_cast<int>((i + 0.5) / prots_per_cell);
    auto rand_nb_cube =
        d_grid->d_cube_id[j] +
        d_nhood[min(static_cast<int>(hiprand_uniform(&d_state[i]) * 27), 26)];
    auto cells_in_cube =
        d_grid->d_cube_end[rand_nb_cube] - d_grid->d_cube_start[rand_nb_cube];
    if (cells_in_cube < 1) return;

    auto a = d_grid->d_point_id[j];
    auto b =
        d_grid->d_point_id[d_grid->d_cube_start[rand_nb_cube] +
                           min(static_cast<int>(
                                   hiprand_uniform(&d_state[i]) * cells_in_cube),
                               cells_in_cube - 1)];
    D_ASSERT(a >= 0);
    D_ASSERT(a < n_cells);
    D_ASSERT(b >= 0);
    D_ASSERT(b < n_cells);
    if (a == b) return;

    if ((d_type[a] != mesenchyme) or (d_type[b] != mesenchyme)) return;

    auto link = &d_link[a * prots_per_cell + i % prots_per_cell];

    auto old_r = d_X[link->a] - d_X[link->b];
    auto old_dist = norm3df(old_r.x, old_r.y, old_r.z);
    auto new_r = d_X[a] - d_X[b];
    auto new_dist = norm3df(new_r.x, new_r.y, new_r.z);
    if (new_dist > r_protrusion) return;

    auto not_initialized = link->a == link->b;
    auto noise = hiprand_uniform(&d_state[i]);
    bool normal_to_w =
        fabs(new_r.w / new_dist) < fabs(old_r.w / old_dist) * (1.f - noise);

    if (not_initialized or normal_to_w) {
        link->a = a;
        link->b = b;
    }
}


int main(int argc, char const* argv[])
{
    // Prepare initial state
    Solution<Cell, Grid_solver> cells{n_max};
    *cells.h_n = n_0;
    random_sphere(0.55f, cells);

    Property<Cell_types> type{n_max};          // translate to Cell_types
    hipMemcpyToSymbol(HIP_SYMBOL(d_type), &type.d_prop, sizeof(d_type));
    for (int i = 0; i < n_0; i++) {
        cells.h_X[i].w = 0.0f;
        type.h_prop[i] = mesenchyme;
    }
    cells.copy_to_device();
    type.copy_to_device();

    Property<int> n_mes_nbs{n_max, "n_mes_nbs"};
    Property<int> n_epi_nbs{n_max, "n_epi_nbs"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_mes_nbs), &n_mes_nbs.d_prop, sizeof(d_mes_nbs));
    hipMemcpyToSymbol(HIP_SYMBOL(d_epi_nbs), &n_epi_nbs.d_prop, sizeof(d_epi_nbs));

    Vtk_output output{"model_features_sequential_addition"};

    // Part 1: Relax a sphere of mesenchymal cells
    for (auto time_step = 0; time_step <= 100; time_step++) {
        cells.copy_to_host();
        n_mes_nbs.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);

        cells.take_step<force, friction_on_background>(dt);

        output.write_positions(cells);
        output.write_polarity(cells);

        output.write_field(cells);
        output.write_property(type);
        output.write_property(n_mes_nbs);
    }

    n_mes_nbs.copy_to_host();
    cells.copy_to_host();
    hipDeviceSynchronize();
    for (int i = 0; i < n_0; i++) {
        if (n_mes_nbs.h_prop[i] < 40){
            auto dist = sqrtf(cells.h_X[i].x * cells.h_X[i].x +
                cells.h_X[i].y * cells.h_X[i].y +
                cells.h_X[i].z * cells.h_X[i].z);
            cells.h_X[i].theta = acosf(cells.h_X[i].z / dist);
            cells.h_X[i].phi = atan2(cells.h_X[i].y, cells.h_X[i].x);
            type.h_prop[i] = epithelium;
        }
    }
    cells.copy_to_device();
    type.copy_to_device();

    // Part 2: Include epithelium surrounding the mesenchyme
    for (auto time_step = 0; time_step <= 100; time_step++) {
        cells.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);

        cells.take_step<force>(dt);

        output.write_positions(cells);
        output.write_polarity(cells);
        output.write_property(type);
        output.write_field(cells);
    }

    cells.copy_to_host();
    hipDeviceSynchronize();
    for (int i = 0; i < n_0; i++) {
        if (cells.h_X[i].x > 1.0f)
            cells.h_X[i].w = 1.0f;
    }
    cells.copy_to_device();

    // Part 3: Add a morphogen gradient.
    for (auto time_step = 0; time_step <= 100; time_step++) {
        cells.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);

        cells.take_step<force>(dt);

        output.write_positions(cells);
        output.write_property(type);
        output.write_field(cells);
    }


    hiprandState* d_state;  // For proliferations
    hipMalloc(&d_state, n_max * sizeof(hiprandState));
    auto seed = time(NULL);
    setup_rand_states<<<(n_max + 128 - 1) / 128, 128>>>(n_max, seed, d_state);

    // Part 4: Add cell proliferation
    for (auto time_step = 0; time_step <= 100; time_step++) {
        cells.copy_to_host();
        type.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);

        cells.take_step<force>(dt);

        proliferate<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
            proliferation_rate, r_min, cells.d_X, cells.d_old_v, cells.d_n,
            d_state);

        output.write_positions(cells);
        output.write_property(type);
        output.write_field(cells);
    }



    Links protrusions{n_max * prots_per_cell, protrusion_strength};
    protrusions.set_d_n(n_0 * prots_per_cell);
    auto intercalation = [&](const int n, const Cell* __restrict__ d_X, Cell* d_dX) {
        return link_forces(protrusions, d_X, d_dX);
    };
    Grid grid{n_max};

    // Part 5: Add cell intercalation. Cells orient their protrusions
    // within the plane perpendicular to the direction of the morphogen
    // gradient.
    for (auto time_step = 0; time_step <= 100; time_step++) {
        cells.copy_to_host();
        protrusions.copy_to_host();
        type.copy_to_host();

        thrust::fill(thrust::device, n_mes_nbs.d_prop,
            n_mes_nbs.d_prop + cells.get_d_n(), 0);
        thrust::fill(thrust::device, n_epi_nbs.d_prop,
            n_epi_nbs.d_prop + cells.get_d_n(), 0);


        protrusions.set_d_n(cells.get_d_n() * prots_per_cell);
        grid.build(cells, r_protrusion);
        update_protrusions<<<(protrusions.get_d_n() + 32 - 1) / 32, 32>>>(
            cells.get_d_n(), grid.d_grid, cells.d_X, protrusions.d_state,
            protrusions.d_link);

        cells.take_step<force>(dt, intercalation);

        output.write_positions(cells);
        output.write_links(protrusions);
        output.write_property(type);
        output.write_field(cells);
    }

    return 0;
}
