#include "hip/hip_runtime.h"
// Toy model for accessing cell fate decisions during cancer development

// Compilation
//
// $ nvcc -std=c++14 -arch=sm_86 {"compiler flags"} Limb_model_simulation.cu
// The values for "-std" and "-arch" flags will depend on your version of CUDA and the specific GPU model you have respectively.
// e.g. -std=c++14 works for CUDA version 11.6 and -arch=sm_86 corresponds to the generation of NVIDIA Geforce 30XX cards.

#include "../include/solvers.cuh"
#include "../include/dtypes.cuh"
#include "../include/inits.cuh"  
#include "../include/property.cuh"
#include "../include/utils.cuh"  
#include "../include/vtk.cuh"

const float r_max = 1.2;                        // Max contact distance between cells
const int n_0 = 500;                           // Initial number of cells
const int n_max = 200000;                       // Max number of cells
const float c_div = 0.0005;                      // Probability of cell division per iteration
const float noise = 0.2;                        // Magnitude of noise returned by generate_noise
const float self_adh = 3.0;                     // Strength of adhesion
const float non_self_rep = 4.0;                 // Strength of repulsion
const float rep_ulim = 0.7;                      // The maximum distance between two cells for which they will repel
const float adh_llim = 0.8;                     // The minimum distance between two cells for which they will atract



const int cont_time = 1000;                  // Simulation duration in arbitrary time units 1000 = 40h ; 750 = 30h
const float dt = 0.1;                           // Time step for Euler integration

// Macro that builds the cell variable type
// MAKE_PT(Cell, u, v); // float3 i .x .y .z .u .v .whatever

__device__ float* d_mechanical_strain; // define global variable for mechanical strain on the GPU (device)
__device__ int* d_cell_type; // global variable for cell type on the GPU
__device__ float3* d_W; // global variable for random number from Weiner process for stochasticity

template<typename Pt>
__device__ Pt pairwise_force(Pt Xi, Pt r, float dist, int i, int j)
{
    Pt dF{0};

    // This will be only useful in simulations with a wall and a ghost node
    if (i == j){
        dF += d_W[i]; // add stochasticity from the weiner process to the attributes of the cells
        return dF;
    }
    if (dist > r_max) return dF;

    float k_adh = (d_cell_type[i] == d_cell_type[j]) ? self_adh : 1.0; // if the cell types are the same set adhesion to 3.0 if not then 1.0
    float k_rep = (d_cell_type[i] == d_cell_type[j]) ? 1.0 : non_self_rep;


    float F = (k_adh * fmaxf(rep_ulim - dist, 0) - k_rep * fmaxf(dist - adh_llim, 0)); // forces are also dependent on adhesion and repulsion between cell types
    // printf("%f\n", F);
    d_mechanical_strain[i] += F; // mechanical strain is the sum of forces on the cell

    dF.x += r.x * F / dist;
    dF.y += r.y * F / dist;
    //dF.z += r.z * F / dist;
    dF.z += 0;

    return dF;
}

__global__ void generate_noise(int n, hiprandState* d_state) { // Weiner process for Heun's method
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    float D = noise; // the magnitude of random noise - set to 0 for deterministic simulation

    // return noise for every attribute of the cell in this case x,y,z
    d_W[i].x = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].y = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    //d_W[i].z = hiprand_normal(&d_state[i]) * powf(dt, 0.5) * D / dt;
    d_W[i].z = 0;
}

__global__ void proliferation(int n_cells, hiprandState* d_state, float3* d_X, float3* d_old_v, int* d_n_cells) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // get the index of the current cell
    if (i >= n_cells) return; // return nothing if the index is greater than n_cells
    if (n_cells >= (n_max * 0.9)) return;

    float rnd = hiprand_uniform(&d_state[i]);
    
    if (rnd > (c_div * dt)) return;

    int n = atomicAdd(d_n_cells, 1);

    float theta = acosf(2. * hiprand_uniform(&d_state[i]) - 1);
    float phi = hiprand_uniform(&d_state[i]) * 2 * M_PI;

    d_X[n].x = d_X[i].x + 0.8 / 4 * sinf(theta) * cosf(phi);
    d_X[n].y = d_X[i].y + 0.8 / 4 * sinf(theta) * sinf(phi);
    d_X[n].z = 0;

    d_old_v[n] = d_old_v[i];

    d_mechanical_strain[n] = 0.0;
    d_cell_type[n] = 1;
}



int main(int argc, char const* argv[])
{

    /*
    Prepare Random Variable for the Implementation of the Wiener Process
    */
    hiprandState* d_state; // define the random number generator on the GPu
    hipMalloc(&d_state, n_max*sizeof(hiprandState)); // allocate GPU memory according to the number of cells
    auto seed = time(NULL); // random number seed - coupled to the time on your machine
    setup_rand_states<<<(n_max + 32 - 1)/32, 32>>>(n_max, seed, d_state); // configuring the random number generator on the GPU (provided by utils.cuh)

    Property<float3> W{n_max, "wiener_process"}; // define a property for the weiner process
    hipMemcpyToSymbol(HIP_SYMBOL(d_W), &W.d_prop, sizeof(d_W)); // connect the global property defined on the GPU to the property defined in this function

    // Initial conditions
    
    Solution<float3, Gabriel_solver> cells{n_max, 50, r_max};
    *cells.h_n = n_0;
    //random_sphere(0.7, cells);
    random_disk_z(0.7, cells);

    cells.copy_to_device();

    // Mechanical strain

    Property<float> mechanical_strain{n_max, "mech_str"}; // create an instance of the property
    hipMemcpyToSymbol(HIP_SYMBOL(d_mechanical_strain), &mechanical_strain.d_prop, sizeof(d_mechanical_strain)); // connect the above instance (on the host) to the global variable on the device

    for (int i = 0; i < n_max; i++) { //initialise with zeroes, for loop step size is set to 1 with i++
        mechanical_strain.h_prop[i] = 0;
    }
    mechanical_strain.copy_to_device();

    auto generic_function = [&](const int n, const float3* __restrict__ d_X, float3* d_dX) { // then set the mechanical forces to zero on the device
        thrust::fill(thrust::device, mechanical_strain.d_prop, mechanical_strain.d_prop + cells.get_d_n(), 0.0);
    };

    // Cell type labels

    Property<int> cell_type{n_max, "cell_type"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_cell_type), &cell_type.d_prop, sizeof(d_cell_type));
    for (int i =0; i < n_0; i++) {
        cell_type.h_prop[i] = std::rand() % 2 + 1; // assign each cell randomly the label 1 or 2
    }
    cell_type.copy_to_device();
        
    Vtk_output output{"relaxation"};

    // Main simulation loop
    for (int time_step = 0; time_step <= cont_time; time_step++) {
        for (float T = 0.0; T < 1.0; T+=dt) {
            proliferation<<<(cells.get_d_n() + 128 - 1)/128, 128>>>(cells.get_d_n(), d_state, cells.d_X, cells.d_old_v, cells.d_n); // simulate proliferation
            generate_noise<<<(cells.get_d_n() + 32 - 1)/32, 32>>>(cells.get_d_n(), d_state); // generate random noise which we will use later on to move the cells
            cells.take_step<pairwise_force, friction_on_background>(dt, generic_function);    
        }

        if(time_step % 10 == 0){
            cells.copy_to_host();
            mechanical_strain.copy_to_host();
            cell_type.copy_to_host();
            output.write_positions(cells);
            output.write_property(mechanical_strain);
            output.write_property(cell_type);
        }
    }
    return 0;
}