#include "hip/hip_runtime.h"
// Toy model for accessing cell fate decisions during cancer development

// Compilation
//
// $ nvcc -std=c++14 -arch=sm_86 {"compiler flags"} Limb_model_simulation.cu
// The values for "-std" and "-arch" flags will depend on your version of CUDA
// and the specific GPU model you have respectively. e.g. -std=c++14 works for
// CUDA version 11.6 and -arch=sm_86 corresponds to the generation of NVIDIA
// Geforce 30XX cards.
#include <thrust/execution_policy.h>
#include <thrust/remove.h>

#include <iterator>

#include "../include/dtypes.cuh"
#include "../include/inits.cuh"
#include "../include/mesh.cuh"
#include "../include/property.cuh"
#include "../include/solvers.cuh"
#include "../include/utils.cuh"
#include "../include/vtk.cuh"
#include "../params/params.h"  // load simulation parameters

// Macro that builds the cell variable type - instead of type float3 we are
// making a instance of Cell with attributes x,y,z,u,v where u and v are
// diffusible chemicals
// MAKE_PT(Cell); // float3 i .x .y .z .u .v .whatever
// to use MAKE_PT(Cell) replace every instance of float3 with Cell
MAKE_PT(Cell, u, v);

// define global variables for the GPU
__device__ float* d_mech_str;
__device__ int* d_cell_type;  // cell_type: A=1-Iri/Mel, B=2-Xan, DEAD=0
__device__ Cell* d_W;  // random number from Weiner process for stochasticity
__device__ bool* d_in_ray;    // whether a cell is in a ray
__device__ Pm d_pm;           // simulation parameters (host h_pm)
__device__ float3 d_tis_min;  // min coordinate of tissue mesh
__device__ float3 d_tis_max;  // max coordinate of tissue mesh
__device__ int* d_ngs_A;      // no. spot cells in neighbourhood
__device__ int* d_ngs_B;      // no. non-spot cells in neighbourhood
__device__ int* d_ngs_Ac;     // overcrowded neighbourhood
__device__ int* d_ngs_Bc;     // overcrowded neighbourhood
__device__ int* d_ngs_Ad;     // donut neighbourhood
__device__ int* d_ngs_Bd;     // donut neighbourhood

template<typename Pt>
__device__ Pt pairwise_force(Pt Xi, Pt r, float dist, int i, int j)
{
    Pt dF{0};

    // counting cells in different nbhds
    if ((dist > 0.318) and (dist < 0.318 + 0.025)) {  // donut
        if (d_cell_type[j] == 1)
            d_ngs_Ad[i] += 1;
        else
            d_ngs_Bd[i] += 1;
    }
    if (dist < 0.075) {  // overcrowding region
        if (d_cell_type[j] == 1)
            d_ngs_Ac[i] += 1;
        else
            d_ngs_Bc[i] += 1;
    }
    if (dist < 0.075) {  // inner disc for cell proliferation conditions
        if (d_cell_type[j] == 1)
            d_ngs_A[i] += 1;
        else
            d_ngs_B[i] += 1;
    }

    if (dist > d_pm.r_max)  // dist = norm3df(r.x, r.y, r.z) solvers line 308
        return dF;          // cutoff for chemical and mechanical interaction

    if (d_cell_type[i] == -1 || d_cell_type[j] == -1 || d_cell_type[i] == -2 ||
        d_cell_type[j] == -2) {
        return dF;  // cells in staging area have no interactions
    }

    if (i == j) {      // if the cell is interacting with itself
        dF += d_W[i];  // add stochasticity from the weiner process to the
                       // attributes of the cells

        // Chemical production and degredation
        if (d_pm.chem_switch) {
            if (d_pm.cmode == 0) {  // chemical production and degredation
                dF.u = d_pm.k_prod * (1.0 - Xi.u) *
                       (d_cell_type[i] == 1 ||
                           d_cell_type[i] == 3);  // cell type 1/3 produce u
                dF.v =
                    d_pm.k_prod * (1.0 - Xi.v) *
                    (d_cell_type[i] == 2);  // cell type 2 produces chemical v
                // dF.u = d_pm.k_prod * ((d_cell_type[i] == 1 || d_cell_type[i]
                // == 3) &
                //                          Xi.u < 1);  // stop making u when it
                //                                      //   reaches 1
                // dF.v = d_pm.k_prod *
                //        ((d_cell_type[i] == 2) & Xi.v < 1);  // stop making v
                //        when
                // it reaches 1
                dF.u -= d_pm.k_deg * (Xi.u);
                dF.v -= d_pm.k_deg * (Xi.v);
            }

            if (d_pm.cmode == 1) {
                // see Schnackenberg 1979 eq. 41
                float a = ((Xi.x + 3) * 0.1);
                float b = ((Xi.y + 1) * 0.2);
                // dF.u = (Xi.u * Xi.u * Xi.v) - Xi.u + d_pm.a_u;
                // dF.v = -(Xi.u * Xi.u * Xi.v) + d_pm.b_v;
                dF.u = (Xi.u * Xi.u * Xi.v) - Xi.u + a;
                dF.v = b - (Xi.u * Xi.u * Xi.v);
                // dF.u = (Xi.u * Xi.u * Xi.v) - Xi.u + (Xi.x * 0.1);
                // dF.v = -(Xi.u * Xi.u * Xi.v) + (Xi.y * 0.1);
            }
            if (d_pm.cmode == 2) {
                // Gray Scott model
                float a = 0.3;
                float b = 0.003;
                float R = 0.1;
                // float a = ((Xi.x + 3) * 0.1);
                // float b = ((Xi.y + 1) * 0.01);
                // float R = ((Xi.x + 3) * 0.1);
                dF.u = R * ((Xi.u * Xi.u * Xi.v) - ((a + b) * Xi.u));
                dF.v = R * (-(Xi.u * Xi.u * Xi.v) + (a * (1 - Xi.v)));
            }
            if (d_pm.cmode == 3) {
                // Gierer Meinhardt model
                // float a = 0.8;
                // float b = 1;
                // float c = 6;
                // float a = ((Xi.x + 3) * 0.1);
                // float b = ((Xi.y + 1) * 0.1);
                // float c = ((Xi.y + 1) * 0.1);
                // dF.u = (a + ((Xi.u * Xi.u) / Xi.v) - (b * Xi.u));
                // dF.v = (Xi.u * Xi.u) - (c * Xi.v);
                const auto lambda = 1;
                const auto f_v = 0.1;
                const auto f_u = 10.0;
                const auto g_u = 5.0;
                const auto m_u = 0.02;
                const auto m_v = 0.05;
                const auto s_u = 0.005;
                dF.u = lambda * ((f_u * Xi.u * Xi.u) / (1 + f_v * Xi.v) -
                                    m_u * Xi.u + s_u);
                dF.v = lambda * (g_u * Xi.u * Xi.u - m_v * Xi.v);
            }
        }
        return dF;
    }

    // Diffusion
    if (d_pm.chem_switch) {
        dF.u = -d_pm.D_u * r.u;  // r = Xi - Xj solvers.cuh line 448
        dF.v = -d_pm.D_v * r.v;
    }
    // dF.u = -((Xi.x + 3) * 0.01) * r.u;
    // dF.v = -((Xi.y + 1.5) * 0.01) * r.v;
    // dF.u = -0.1 * r.u;
    // dF.v = -4 * r.v;
    // dF.u = -Xi.x * r.u * 0.01;
    // dF.v = -Xi.y * r.v * 0.01;
    // dF.u = -1 * r.u;
    // dF.v = -40 * r.v;
    // dF.u = -0.01 * r.u;
    // dF.v = -0.05 * r.v;

    // Mechanical forces

    if (!d_pm.mov_switch)
        return dF;  // if cell movement is off, return no forces

    // default adhesion and repulsion vals for cell interactions
    float Adh = 0;  // d_pm.Add;
    float adh = 0;  // d_pm.add;
    float Rep = 0;  // d_pm.Rdd;
    float rep = 0;  // d_pm.rdd;

    if (d_pm.diff_adh_rep) {
        if (d_cell_type[i] == 1 and d_cell_type[j] == 1) {
            Adh = 0;  // A-A interact with different adh and rep vals
            adh = 1;
            Rep = 0.00124;
            rep = 0.02;
        }
        if (d_cell_type[i] == 2 and d_cell_type[j] == 1) {
            Adh = 0;  // A-A interact with different adh and rep vals
            adh = 1;
            Rep = 0.00274;
            rep = 0.02;
        }
        if (d_cell_type[i] == 1 and d_cell_type[j] == 2) {
            Adh = 0.001956;  // B-B interact with different adh and rep vals
            adh = 0.012;
            Rep = 0.00226;
            rep = 0.02;
        }
        if (d_cell_type[i] == 2 and d_cell_type[j] == 2) {
            Adh = 0;  // A-B interact with different adh and rep vals
            adh = 1;
            Rep = 0.00055;
            rep = 0.011;
        }
    }


    // float F = (k_rep * fmaxf(0.08 - dist, 0) - k_adh * fmaxf(dist - 0.08,
    // 0)); // forces are also dependent on adhesion and repulsion between cell
    // types float F = (Adh * r.x * exp(-sqrt(r.x^2 + r.y^2) / adh)) / (adh *
    // sqrt(r.x^2 + r.y^2)) - (Rep * r.x * exp(-sqrt(r.x^2 - r.y^2) / rep) /
    // (rep * sqrt(r.x^2 - r.y^2))); Volkening et al. 2015 force potential,
    // function in terms of distance in n dimensions
    float term1 = Adh / adh * expf(-dist / adh);
    float term2 = Rep / rep * expf(-dist / rep);
    float F = term1 - term2;
    // printf("%f\n", F);
    d_mech_str[i] -= F;  // mechanical strain is the sum of forces on the cell

    dF.x -= r.x * F / dist;
    dF.y -= r.y * F / dist;
    dF.z -= 0;

    // dF is the change in x,y,z,u,v etc. over dt, for a particular pairwise
    // interaction. Yalla sums the dFs for all interactions for cell i to give
    // d_dX[i] Yalla compute the new values by multiplying d_dX[i] by dt and
    // adding to the values in the current time step This function is in solvers
    // in the euler_step function

    return dF;
}

__global__ void generate_noise(int n, hiprandState* d_state)
{  // Weiner process for Heun's method
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    // return noise for every attribute of the cell in this case x,y,z
    d_W[i].x =
        hiprand_normal(&d_state[i]) * powf(d_pm.dt, 0.5) * d_pm.noise / d_pm.dt;
    d_W[i].y =
        hiprand_normal(&d_state[i]) * powf(d_pm.dt, 0.5) * d_pm.noise / d_pm.dt;
    d_W[i].z = 0;
    d_W[i].u = 0;
    d_W[i].v = 0;
}


__global__ void stage_new_cells(int n_cells, hiprandState* d_state, Cell* d_X,
    float3* d_old_v, int* d_n_cells)
{
    int i = blockIdx.x * blockDim.x +
            threadIdx.x;  // get the index of the current cell
    if (i >= n_cells)
        return;  // return nothing if the index is greater than n_cells
    if (n_cells >= (d_pm.n_max * 0.9))
        return;  // return nothing if the no. cells starts to approach the max

    if (i < d_pm.n_new_cells) {  // threads with i < n_new_cell create new cell
        int n = atomicAdd(d_n_cells, 1);
        d_X[n].x = d_tis_min.x +
                   (d_tis_max.x - d_tis_min.x) * hiprand_uniform(&d_state[i]);
        d_X[n].y = d_tis_min.y +
                   (d_tis_max.y - d_tis_min.y) * hiprand_uniform(&d_state[i]);
        d_X[n].z = 0;

        d_old_v[n] = d_old_v[i];
        // d_cell_type[n] = -1;
        if (i < (d_pm.n_new_cells / 2)) {  // stage 1/2 cells of each type
            d_cell_type[n] = -1;
        } else {
            d_cell_type[n] = -2;
        }
    }
}

__global__ void clean_up(int n_cells, Cell* d_X, int* d_n_cells)
{
    // Remove cells that are marked for death by swapping with last cell.
    // N.B. if n-1 is also dead, a dead cell will remain until the next call,
    // thus this function is called repeatedly until no dead cells remain.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) return;

    if (d_cell_type[i] == -1 || d_cell_type[i] == -2) {
        int n = atomicSub(d_n_cells, 1);  // decrement d_n_cells
        if (i < n) {
            d_X[i] = d_X[n - 1];  // copy properties of last cell to cell i
            d_W[i] = d_W[n - 1];
            d_cell_type[i] = d_cell_type[n - 1];
            d_mech_str[i] = d_mech_str[n - 1];
            // d_old_v[i] = d_old_v[n - 1];
            d_in_ray[i] = d_in_ray[n - 1];
            d_ngs_A[i] = d_ngs_A[n - 1];
            d_ngs_B[i] = d_ngs_B[n - 1];
            d_ngs_Ac[i] = d_ngs_Ac[n - 1];
            d_ngs_Bc[i] = d_ngs_Bc[n - 1];
            d_ngs_Ad[i] = d_ngs_Ad[n - 1];
            d_ngs_Bd[i] = d_ngs_Bd[n - 1];
        }
    }
}

__global__ void proliferation(int n_cells, hiprandState* d_state, Cell* d_X,
    float3* d_old_v, int* d_n_cells)
{
    // change cells from staging to active types if conditions are met
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // index of current cell

    if (i >= n_cells) return;                   // stop if i >= n_cells
    if (n_cells >= (d_pm.n_max * 0.9)) return;  // no div above n_max
    // if (d_mech_str[i] > d_pm.mech_thresh) return;   // no div above

    // if (d_cell_type[i] == 1 || d_cell_type[i] == 2) return;

    if (d_cell_type[i] == -1) {
        if (d_ngs_A[i] > d_pm.alpha * d_ngs_B[i] &&   // short range
            d_ngs_Bd[i] > d_pm.beta * d_ngs_Ad[i] &&  // long range
            d_ngs_Ac[i] + d_ngs_Bc[i] < d_pm.eta      // overcrowding

        )
            d_cell_type[i] = 1;
    } else if (d_cell_type[i] == -2) {
        if (d_ngs_B[i] > d_pm.phi * d_ngs_A[i] &&    // short range
            d_ngs_Ad[i] > d_pm.psi * d_ngs_Bd[i] &&  // long range
            d_ngs_Ac[i] + d_ngs_Bc[i] < d_pm.kappa   // overcrowding
        )
            d_cell_type[i] = 2;
    }
}

__global__ void death(
    int n_cells, hiprandState* d_state, Cell* d_X, int* d_n_cells)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) return;
    float r = hiprand_uniform(&d_state[i]);
    if (d_cell_type[i] == -1 || d_cell_type[i] == -2) return;  // don't die

    // short-range competition spot cells
    // if spot and non-spot in nbhd exceed spot, die
    if (d_cell_type[i] == 1 and d_ngs_B[i] > d_ngs_A[i]) {
        d_cell_type[i] = -1;  // mark for death
    }
    // short-range competition non-spot cells
    // if non-spot and no. spot in nbhd  exceeds no. non-spot, die
    if (d_cell_type[i] == 2 and d_ngs_A[i] > d_ngs_B[i]) {
        d_cell_type[i] = -2;
    }
    // long range spot-cell death condition
    // if spot and no. spot in donut exceeds no. non-spot, die
    if (d_cell_type[i] == 1 and d_ngs_Ad[i] > d_pm.xi * d_ngs_Bd[i] and
        (r > d_pm.q_death)) {
        d_cell_type[i] = -1;
    }
}

__global__ void cell_switching(int n_cells, Cell* d_X)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) return;

    // spot cells become static when u is high
    // if (d_cell_type[i] == 1 && d_X[i].u > 0.5) d_cell_type[i] = 3;
    // if (d_pm.tmode == 5) {  // switching for non-advecting/advecting spot
    // cells
    //     float top_y = d_tis_max.y - (0.4 * (d_tis_max.y - d_tis_min.y));
    //     float bot_y = d_tis_min.y + (0.4 * (d_tis_max.y - d_tis_min.y));
    //     if (d_cell_type[i] == 1 && d_X[i].u > 0.5 && d_X[i].y < top_y &&
    //         d_X[i].y > bot_y)
    //         d_cell_type[i] = 3;  // don't switch if still in top 10% of
    //         tissue
    // }
    // if (d_cell_type[i] == 2 && d_X[i].u > 180) {
    //     d_cell_type[i] = 1;  // switch to spot cell if u high
    // }
    // if (d_cell_type[i] == 1 && d_X[i].u < 180) {
    //     d_cell_type[i] = 2;  // switch to non-spot cell if u low
    // }
    float top_y = d_tis_max.y - (0.2 * (d_tis_max.y - d_tis_min.y));
    float bot_y = d_tis_min.y + (0.2 * (d_tis_max.y - d_tis_min.y));
    if (d_X[i].y < top_y && d_X[i].y > bot_y) {
        if (d_cell_type[i] == 1 && d_X[i].v > d_pm.vthresh) {
            d_cell_type[i] = 3;  // switch to spot cell if u high
        }
        if (d_cell_type[i] == 3 && d_X[i].v < d_pm.vthresh) {
            d_cell_type[i] = 2;  // switch to non-spot cell if u low
        }
    }
}


void init_rays(Mesh& tis, float rays[100][2])  // maximum of 100 rays
{
    // host function for initialising rays
    // float rays[n_ray][2];  // start and end of each ray
    float p_min, p_max;
    if (h_pm.ray_dir == 0) {
        p_min = tis.get_minimum().x;
        p_max = tis.get_maximum().x;
    }
    if (h_pm.ray_dir == 1) {
        p_min = tis.get_minimum().y;
        p_max = tis.get_maximum().y;
    }
    float step;
    step = (p_max - p_min) / (h_pm.n_rays - 1);
    if (h_pm.n_rays < 2)
        step = (p_max - p_min) / 2;  // if only one ray, set to middle

    for (int i = 0; i < h_pm.n_rays; i++) {
        float p1 = p_min + i * step;  // start of ray either x or y line
        float p2 = p1 + (h_pm.s_ray * (p_max - p_min));  // scale by tissue size
        // x_pairs.push_back({x1, x2});
        rays[i][0] = p1;
        rays[i][1] = p2;
    }
}

__global__ void advection(int n_cells, const Cell* d_X, Cell* d_dX,
    const float (*rays)[2], int time_step)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_cells) return;

    if (d_cell_type[i] == 1) {  // only type 1 cells advect
        float norm_x = (d_X[i].x - d_tis_min.x) /
                       (d_tis_max.x - d_tis_min.x);  // % along x axis
        float ad_time =
            norm_x * d_pm.cont_time * 0.5;  // wait time proportional to norm_x

        float ad = d_pm.ad_s;  // default advection strength
        if (d_pm.ad_func == 1 && time_step < ad_time) ad = 0;

        if (d_pm.ray_switch) {
            for (int k = 0; k < d_pm.n_rays; ++k) {
                d_in_ray[i] = false;
                float pos;
                if (d_pm.ray_dir == 0) pos = d_X[i].x;
                if (d_pm.ray_dir == 1) pos = d_X[i].y;
                if (pos >= rays[k][0] && pos <= rays[k][1]) {
                    d_in_ray[i] = true;
                    ad = d_pm.soft_ad_s;  // soft_ad if in ray
                    if (d_pm.ad_func == 1 && time_step < ad_time) ad = 0;
                    break;  // A-P time delay
                }
            }
        }
        if (d_pm.ad_dir == 0) d_dX[i].x += ad;  // +ve x direction
        if (d_pm.ad_dir == 1) d_dX[i].y -= ad;  // -ve y direction
    }
}

__device__ bool is_dead_type(int type) { return type == -1 || type == -2; }

int tissue_sim(int argc, char const* argv[], int walk_id = 0, int step = 0)
{
    std::cout << std::fixed
              << std::setprecision(6);  // set precision for floats
    // h_pm.dt = 0.05 * 0.6 * 0.6 / h_pm.D_v;

    // Prepare Random Variable for the Implementation of the Wiener Process
    hiprandState* d_state;  // define the random number generator on the GPu
    hipMalloc(&d_state,
        h_pm.n_max * sizeof(hiprandState));  // allocate GPU memory according
                                            // to no. cells
    auto seed =
        time(NULL);  // random number seed - coupled to the time on your machine
    setup_rand_states<<<(h_pm.n_max + 32 - 1) / 32, 32>>>(h_pm.n_max, seed,
        d_state);  // configuring the random number generator
                   // on the GPU (provided by utils.cuh)

    /* create host variables*/
    // you first create an instance of the Property class on the host, then
    // you connect it to the global variable defined on the device with
    Property<Cell> W{
        h_pm.n_max, "wiener_process"};  // weiner process random number
    hipMemcpyToSymbol(HIP_SYMBOL(d_W), &W.d_prop, sizeof(d_W));
    Property<float> mech_str{h_pm.n_max, "mech_str"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_mech_str), &mech_str.d_prop, sizeof(d_mech_str));
    Property<int> cell_type{h_pm.n_max, "cell_type"};  // cell type labels
    hipMemcpyToSymbol(HIP_SYMBOL(d_cell_type), &cell_type.d_prop, sizeof(d_cell_type));
    Property<bool> in_ray{h_pm.n_max, "in_ray"};  // whether cell in ray or not
    hipMemcpyToSymbol(HIP_SYMBOL(d_in_ray), &in_ray.d_prop, sizeof(d_in_ray));
    hipMemcpyToSymbol(HIP_SYMBOL(d_pm), &h_pm, sizeof(Pm));  // copy host params
    Property<int> ngs_A{h_pm.n_max, "ngs_A"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_A), &ngs_A.d_prop, sizeof(d_ngs_A));
    Property<int> ngs_B{h_pm.n_max, "ngs_B"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_B), &ngs_B.d_prop, sizeof(d_ngs_B));
    Property<int> ngs_Ac{h_pm.n_max, "ngs_Ac"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_Ac), &ngs_Ac.d_prop, sizeof(d_ngs_Ac));
    Property<int> ngs_Bc{h_pm.n_max, "ngs_Bc"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_Bc), &ngs_Bc.d_prop, sizeof(d_ngs_Bc));
    Property<int> ngs_Ad{h_pm.n_max, "ngs_Ad"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_Ad), &ngs_Ad.d_prop, sizeof(d_ngs_Ad));
    Property<int> ngs_Bd{h_pm.n_max, "ngs_Bd"};
    hipMemcpyToSymbol(HIP_SYMBOL(d_ngs_Bd), &ngs_Bd.d_prop, sizeof(d_ngs_Bd));

    // Initial conditions
    // Solution<Cell, Gabriel_solver> cells{h_pm.n_max, h_pm.g_size,
    // h_pm.r_max};
    Solution<Cell, Grid_solver> cells{h_pm.n_max, h_pm.g_size, h_pm.r_max * 5};
    // args are n_max, grid_size, cube_size
    // *cells.h_n = h_pm.n_0;

    float rays[h_pm.n_rays][2];  // initialise rays with default values
    for (int i = 0; i < h_pm.n_rays; i++) {
        rays[i][0] = 0;
        rays[i][1] = 0;
    }
    // Allocate memory for rays on the device
    float (*d_rays)[2];
    hipMalloc(&d_rays, h_pm.n_rays * 2 * sizeof(float));

    if (h_pm.tmode == 0) {
        random_disk_z(h_pm.init_dist, cells);
        for (int i = 0; i < h_pm.n_0; i++) {
            cell_type.h_prop[i] = (std::rand() % 100 < h_pm.A_init)
                                      ? 1
                                      : 2;  // randomly assign a proportion of
                                            // initial cells with each type
        }
    }
    if (h_pm.tmode == 1) {
        regular_rectangle(h_pm.init_dist,
            std::round(std::sqrt(h_pm.n_0) / 10) * 10,
            cells);  // initialise rectangle specifying the no. cells along
                     // the x axis
        for (int i = 0; i < h_pm.n_0; i++) {
            cell_type.h_prop[i] = (std::rand() % 100 < h_pm.A_init) ? 1 : 2;
        }
    }
    if (h_pm.tmode == 2) {  // rectangle with spots on one end
        auto sp_size =
            (h_pm.A_init / 100.0) * h_pm.n_0;  // calculate no. cells in spot
        regular_rectangle_w_spot(sp_size, h_pm.init_dist,
            std::round(std::sqrt(h_pm.n_0) / 10) * 10, cells);
        for (int i = 0; i < h_pm.n_0; i++) {
            cell_type.h_prop[i] = (i < h_pm.n_0 - sp_size)
                                      ? 2
                                      : 1;  // set cell type to 1 for spot
                                            // cells, and 2 for all others
        }
    }
    if (h_pm.tmode ==
        3) {  // cut the tissue mesh out of a random cloud of cells
        Mesh tis{"../inits/shape1_mesh_3D.vtk"};
        tis.rescale(h_pm.tis_s);  // expand the mesh to fit to the boundaries
        auto tis_min = tis.get_minimum();
        auto tis_max = tis.get_maximum();
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_min), &tis_min, sizeof(float3));  // tis min
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_max), &tis_max, sizeof(float3));  // tis max
        random_rectangle(
            h_pm.init_dist, tis.get_minimum(), tis.get_maximum(), cells);
        auto new_n =
            thrust::remove_if(thrust::host, cells.h_X, cells.h_X + *cells.h_n,
                [&tis](Cell x) { return tis.test_exclusion(x); });
        *cells.h_n = std::distance(cells.h_X, new_n);
        for (int i = 0; i < h_pm.n_0; i++) {  // set cell types
            cell_type.h_prop[i] = (std::rand() % 100 < h_pm.A_init)
                                      ? 1
                                      : 2;  // set cell type to 1 for spot
                                            // cells, and 2 for all others
        }
    }
    if (h_pm.tmode == 4) {  // cut the fin mesh out of a random cloud of cells
        Mesh tis{"../inits/shape1_mesh_3D.vtk"};
        tis.rescale(h_pm.tis_s);
        auto tis_min = tis.get_minimum();
        auto tis_max = tis.get_maximum();
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_min), &tis_min, sizeof(float3));  // tis min
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_max), &tis_max, sizeof(float3));  // tis max
        auto x_len = tis.get_maximum().x - tis.get_minimum().x;
        random_rectangle(
            h_pm.init_dist, tis.get_minimum(), tis.get_maximum(), cells);
        auto new_n =
            thrust::remove_if(thrust::host, cells.h_X, cells.h_X + *cells.h_n,
                [&tis](Cell x) { return tis.test_exclusion(x); });
        *cells.h_n = std::distance(cells.h_X, new_n);
        for (int i = 0; i < h_pm.n_0; i++) {  // set cell types
            // spot cells appear in leftmost 10% of tissue
            if (cells.h_X[i].x < tis.get_minimum().x + (x_len * 0.1))
                cell_type.h_prop[i] = (std::rand() % 100 < 50) ? 1 : 2;
            else
                cell_type.h_prop[i] = 2;
        }
        init_rays(tis, rays);
        // Print the values of rays after initialization
        std::cout << "Rays after initialization:" << std::endl;
        for (int i = 0; i < h_pm.n_rays; i++) {
            std::cout << "Ray " << i << ": (" << rays[i][0] << ", "
                      << rays[i][1] << ")" << std::endl;
        }
    }
    if (h_pm.tmode == 5) {  // fin with spot aggregation at top
        Mesh tis{"../inits/shape1_mesh_3D.vtk"};
        tis.rescale(h_pm.tis_s);
        auto tis_min = tis.get_minimum();
        auto tis_max = tis.get_maximum();
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_min), &tis_min, sizeof(float3));  // tis min
        hipMemcpyToSymbol(HIP_SYMBOL(d_tis_max), &tis_max, sizeof(float3));  // tis max
        auto y_len = tis.get_maximum().y - tis.get_minimum().y;
        random_rectangle(
            h_pm.init_dist, tis.get_minimum(), tis.get_maximum(), cells);
        auto new_n =
            thrust::remove_if(thrust::host, cells.h_X, cells.h_X + *cells.h_n,
                [&tis](Cell x) { return tis.test_exclusion(x); });
        *cells.h_n = std::distance(cells.h_X, new_n);
        for (int i = 0; i < h_pm.n_0; i++) {  // set cell types
            // spot cells appear in topmost 10% of tissue
            if (cells.h_X[i].y > tis.get_maximum().y - (y_len * 0.1))
                cell_type.h_prop[i] = (std::rand() % 100 < 50) ? 1 : 2;
            else
                cell_type.h_prop[i] = 2;
        }
        init_rays(tis, rays);
        // Print the values of rays after initialization
        std::cout << "Rays after initialization:" << std::endl;
        for (int i = 0; i < h_pm.n_rays; i++) {
            std::cout << "Ray " << i << ": (" << rays[i][0] << ", "
                      << rays[i][1] << ")" << std::endl;
        }
    }


    for (int i = 0; i < h_pm.n_0; i++) {  // initialise chemical amounts
        // cells.h_X[i].u = (std::rand()) / (RAND_MAX + 1.);
        // cells.h_X[i].v = (std::rand()) / (RAND_MAX + 1.);
        cells.h_X[i].u = 0;
        cells.h_X[i].v = 0;
        // Mesh tis{"../inits/shape1_mesh_3D.vtk"};
        // tis.rescale(h_pm.tis_s);
        // auto y_len = tis.get_maximum().y - tis.get_minimum().y;
        // if (cells.h_X[i].y > tis.get_maximum().y - (y_len * 0.1)) {
        //     cells.h_X[i].u = (std::rand()) / (RAND_MAX + 1.);
        //     cells.h_X[i].v = (std::rand()) / (RAND_MAX + 1.);
        // }
    }

    // Initialise properties and k with zeroes
    for (int i = 0; i < h_pm.n_max; i++) {  // initialise with zeroes
        mech_str.h_prop[i] = 0;
        in_ray.h_prop[i] = false;
        ngs_A.h_prop[i] = 0;
        ngs_B.h_prop[i] = 0;
        ngs_Ac.h_prop[i] = 0;
        ngs_Bc.h_prop[i] = 0;
        ngs_Ad.h_prop[i] = 0;
        ngs_Bd.h_prop[i] = 0;
    }

    // Copy the ray data to the device
    hipMemcpy(
        d_rays, &rays, h_pm.n_rays * 2 * sizeof(float), hipMemcpyHostToDevice);

    int time_step;  // declare  outside main loop for access in gen_func
    auto generic_function = [&](const int n, const Cell* __restrict__ d_X,
                                Cell* d_dX) {  // then set the mechanical
        // forces to zero on the device
        // remove cells marked for death
        // auto new_end = thrust::remove_if(thrust::device,
        // cell_type.d_prop,
        //     cell_type.d_prop + n_cells,
        //     [] __device__(int type) { return type == -1 || type == -2;
        //     });
        // n_cells = new_end - d_X;
        // Set these properties to zero after every timestep so they
        // don't accumulate Called every timesetep, allows you to add
        // custom forces at every timestep e.g. advection
        thrust::fill(thrust::device, mech_str.d_prop,
            mech_str.d_prop + cells.get_d_n(), 0.0);
        thrust::fill(thrust::device, in_ray.d_prop,
            in_ray.d_prop + cells.get_d_n(), false);
        thrust::fill(
            thrust::device, ngs_A.d_prop, ngs_A.d_prop + cells.get_d_n(), 0);
        thrust::fill(
            thrust::device, ngs_B.d_prop, ngs_B.d_prop + cells.get_d_n(), 0);
        thrust::fill(
            thrust::device, ngs_Ac.d_prop, ngs_Ac.d_prop + cells.get_d_n(), 0);
        thrust::fill(
            thrust::device, ngs_Bc.d_prop, ngs_Bc.d_prop + cells.get_d_n(), 0);
        thrust::fill(
            thrust::device, ngs_Ad.d_prop, ngs_Ad.d_prop + cells.get_d_n(), 0);
        thrust::fill(
            thrust::device, ngs_Bd.d_prop, ngs_Bd.d_prop + cells.get_d_n(), 0);

        // return wall_forces<Cell, boundary_force>(n, d_X, d_dX, 0);
        if (h_pm.adv_switch)
            advection<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                cells.get_d_n(), d_X, d_dX, d_rays, time_step);
        if (h_pm.fin_walls)
            return wall_forces_mult<Cell, boundary_forces_mult>(n, d_X, d_dX, 0,
                h_pm.w_off_s);  //, num_walls, wall_normals, wall_offsets);
    };

    cells.copy_to_device();
    mech_str.copy_to_device();
    cell_type.copy_to_device();
    in_ray.copy_to_device();
    ngs_A.copy_to_device();
    ngs_B.copy_to_device();
    ngs_Ac.copy_to_device();
    ngs_Bc.copy_to_device();
    ngs_Ad.copy_to_device();
    ngs_Bd.copy_to_device();

    Vtk_output output{
        "out_" + std::to_string(walk_id) + "_" + std::to_string(step)};
    // create instance of Vtk_output class


    /* the neighbours are initialised with 0. However, you want to use them
       in the proliferation function, which is called first.
        1. proliferation
        2. noise
        3. take_step
       we use a trick, such that the very first call of the proliferation is
       not launched on zeros. here instead of dt we pass 0.0, so that we
       count cells, but do not compute any replacements in the tissue
       -> x[t+1] = x[t] + 0.0 * (dx);
    */

    cells.take_step<pairwise_force>(0.0, generic_function);

    // write out initial condition
    cells.copy_to_host();
    mech_str.copy_to_host();
    cell_type.copy_to_host();
    in_ray.copy_to_host();
    ngs_A.copy_to_host();
    ngs_B.copy_to_host();
    ngs_Ac.copy_to_host();
    ngs_Bc.copy_to_host();
    ngs_Ad.copy_to_host();
    ngs_Bd.copy_to_host();

    output.write_positions(cells);
    output.write_property(mech_str);
    output.write_property(cell_type);
    output.write_property(in_ray);
    // output.write_field(cells, "u", &Cell::u);  // write u of each cell to
    // vtk output.write_field(cells, "v", &Cell::v);
    output.write_property(ngs_A);
    output.write_property(ngs_B);
    output.write_property(ngs_Ac);
    output.write_property(ngs_Bc);
    output.write_property(ngs_Ad);
    output.write_property(ngs_Bd);


    // Main simulation loop
    for (time_step = 0; time_step <= h_pm.cont_time; time_step++) {
        for (float T = 0.0; T < 1.0; T += h_pm.dt) {
            // printf("T = %f\n", T);
            generate_noise<<<(cells.get_d_n() + 32 - 1) / 32, 32>>>(
                cells.get_d_n(),
                d_state);  // generate random noise which we will use later
                           // on to move the cells
            if (h_pm.prolif_switch) {
                if (time_step % int(h_pm.cont_time / 500) == 0) {
                    stage_new_cells<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                        cells.get_d_n(), d_state, cells.d_X, cells.d_old_v,
                        cells.d_n);  // stage new cells
                }
                cells.take_step<pairwise_force>(0.0, generic_function);
                proliferation<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                    cells.get_d_n(), d_state, cells.d_X, cells.d_old_v,
                    cells.d_n);  // simulate proliferation
                // clean_up<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                //     cells.get_d_n(), cells.d_X, cells.d_n);  // remove cells
            }
            if (h_pm.type_switch)
                cell_switching<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                    cells.get_d_n(), cells.d_X);  // switch cell types if
            // conditions are met


            cells.take_step<pairwise_force, friction_on_background>(
                h_pm.dt, generic_function);
            if (h_pm.death_switch)  // death occurs once per day - 20 days total
                if (time_step % int(h_pm.cont_time / 20) == 0) {
                    death<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                        cells.get_d_n(), d_state, cells.d_X, cells.d_n);
                }
            int prev_n, curr_n;
            // Remove cells marked for death, repeat until all removed
            do {
                prev_n = cells.get_d_n();
                clean_up<<<(cells.get_d_n() + 128 - 1) / 128, 128>>>(
                    cells.get_d_n(), cells.d_X, cells.d_n);
                curr_n = cells.get_d_n();
            } while (curr_n < prev_n);  // Repeat until cell count stabilizes
        }

        if (time_step % int(h_pm.cont_time / h_pm.no_frames) == 0) {
            cells.copy_to_host();
            mech_str.copy_to_host();
            cell_type.copy_to_host();
            in_ray.copy_to_host();
            ngs_A.copy_to_host();
            ngs_B.copy_to_host();
            ngs_Ac.copy_to_host();
            ngs_Bc.copy_to_host();
            ngs_Ad.copy_to_host();
            ngs_Bd.copy_to_host();


            output.write_positions(cells);
            output.write_property(mech_str);
            output.write_property(cell_type);
            output.write_property(in_ray);
            // output.write_field(cells, "u", &Cell::u);
            // output.write_field(cells, "v", &Cell::v);
            output.write_property(ngs_A);
            output.write_property(ngs_B);
            output.write_property(ngs_Ac);
            output.write_property(ngs_Bc);
            output.write_property(ngs_Ad);
            output.write_property(ngs_Bd);
        }
    }
    return 0;
}

// compile tissue_sim as main when this file is not included as library
// elsewhere
#ifndef COMPILE_AS_LIBRARY
int main(int argc, char const* argv[]) { return tissue_sim(argc, argv); }
#endif